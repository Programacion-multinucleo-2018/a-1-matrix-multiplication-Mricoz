#include "hip/hip_runtime.h"
/*
    Matrix Multiplication on GPU
*/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <math.h>

#include "common.h"

#define SIZE 4000

////////// FUNCTION DECLARATIONS //////////

void printMatrix(int * M);
void fillMatrix(int * M);
//bool checkResult(int * m_host, int * m_gpu);
//void checkResult2(int *hostRef, int *gpuRef);
//void matrixMultHost(int * C, int * A, int * B);
__global__ void matrixMultGPU(int * A, int * B, int * C);

////////// FUNCTION DEFINITIONS //////////

//Print the matrix
void printMatrix(int * M){
    int size = SIZE * SIZE;
    for (int i = 0; i < size; i++) {
        std::cout << M[i] << " ";
    }
    std::cout << "\n";
}

// Function to fill the matrix
void fillMatrix(int * M){
    int size = SIZE * SIZE;
    // Fill the matrices with random numbers 1-10
    for (int i = 0; i < size; i++) {
        M[i] = rand() % 10 + 1;
    }
}

// // Check that the two matrices match
// bool checkResult(int * m_host, int * m_gpu){
//     int size = SIZE * SIZE;
//     for(int i = 0; i< size; i++){
//         if(m_host[i]!= m_gpu[i]){
//             return false;
//         }
//     }
//     return true;
// }

// // Function to multiply matrices cpu (Host)
// void matrixMultHost(int * C, int * A, int * B){
//     for(int i = 0; i < SIZE; i++){
//         for(int j = 0; j < SIZE; j++){
//             for(int k = 0; k < SIZE; k++){
//                 C[i * SIZE + j] += A[k + i * SIZE] * B[k * SIZE + j];
//             }
//         }
//     }
// }

// Function to multiply matrices gpu
__global__ void matrixMultGPU(int * A, int * B, int * C){

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = ix * SIZE + iy;

    int temp = 0;

    if (ix < SIZE && iy < SIZE){
        for(int i = 0; i < SIZE ; i++){
            temp += A[ix * SIZE + i] * B[i * SIZE + iy];
        }
        C[idx] = temp;
    }
}

// Main function
int main(int argc, char **argv){

    srand(time(NULL));

    std::cout << "\n";
    std::cout << "---------- MATRIX MULTIPLICATION CPU (NO THREADS) ----------" << "\n";
    std::cout << "\n";

    std::cout << "initializing..." << "\n\n";

    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // Set up data size of matrix
    int nx = SIZE;
    int ny = SIZE;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);
    std::cout << "Matrix size: nx " << nx << " ny " << ny << "\n";

    // Malloc host memory
    int *A, *B, *gpuRef;
    A = (int *)malloc(nBytes);
    B = (int *)malloc(nBytes);
    //hostRef = (int *)malloc(nBytes);
    gpuRef = (int *)malloc(nBytes);

    // Initialize data at host side

    fillMatrix(A);
    fillMatrix(B);

    //printMatrix(A);
    //printMatrix(B);

    // Fill with 0
    //memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // Multiply matrix at host side for result SAFE_CALLs
    // auto start_cpu = std::chrono::high_resolution_clock::now();
    // matrixMultHost(hostRef, A, B);
    // auto end_cpu = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
    //
    // std::cout << "matrixMultHost elapsed " << duration_ms.count() << " ms" << "\n";

    // malloc device global memory
    int *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side
    int dimx = 1;
    int dimy = 128;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    auto start_cpu =  std::chrono::high_resolution_clock::now();
    matrixMultGPU<<<grid, block>>>(d_MatA, d_MatB, d_MatC);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    auto end_cpu =  std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("matrixMultGPU <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x, grid.y, block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // Compare results
    //checkResult(hostRef, gpuRef);

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(A);
    free(B);
    //free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
